#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


extern "C" {

    __global__ void initCurand(hiprandState* states, uint32_t seed, uint32_t nElem ) {
	    uint32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
	    if (tid < nElem) {
	        states[tid] = hiprandState();
	        hiprand_init( seed, tid, 0, &states[tid] );
	    }
    }

    __global__ void initColoring(uint32_t nnodes, uint32_t * coloring_d, float nCol, hiprandState * states, uint32_t seed) {

	    uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	    if (idx >= nnodes)
		    return;

	    float randnum = hiprand_uniform(&states[idx]);

	    int color = (int)(randnum * nCol);
	    //printf("color=%d\n", states[idx].d);

	    coloring_d[idx] = color;
	    //coloring_d[idx] = 0;
    }

    __global__ void conflictChecker(uint32_t nedges, uint32_t * conflictCounter_d, uint32_t * coloring_d, uint32_t * edges) {

        uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

        if (idx >= nedges)
            return;

        uint32_t idx0 = idx * 2;
        uint32_t idx1 = idx0 + 1;

        uint32_t node0 = edges[idx0];
        uint32_t node1 = edges[idx1];

        uint32_t col0 = coloring_d[node0];
        uint32_t col1 = coloring_d[node1];

        conflictCounter_d[idx] = col0 == col1;
    }
}

