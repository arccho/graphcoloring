#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

extern "C" {
    __global__ void testRand( hiprandState * state, int nb ){
        int id = threadIdx.x  + blockIdx.x * blockDim.x;
        int value;
        for (int i=0;i<nb;i++){
            hiprandState localState = state[id];
            value = hiprand(&localState);
            //state[id] = localState;
            printf("Id %i, value %i\n",id,value);
        }
    }
    __global__ void setup_kernel( hiprandState * state, unsigned long seed )
    {
        int id = threadIdx.x  + blockIdx.x * blockDim.x;
        hiprand_init( seed, id , 0, &state[id] );
    }
}