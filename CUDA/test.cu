
#include <hip/hip_runtime.h>
template <class T>
__device__ T incr(T x) {
    return (x + 1.0);
}

// Needed to avoid name mangling so that PyCUDA can
// find the kernel function:
extern "C" {
    __global__ void func(float *a, int N)
    {
        int idx = threadIdx.x;
        if (idx < N)
            a[idx] = incr(a[idx]);
    }
}