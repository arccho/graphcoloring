// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#ifdef WIN32
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#endif

#include "GPUStream.h"

GPUStream::GPUStream( uint32_t n ) : numThreads( n ) {

    streams = new hipStream_t[numThreads];

    for (uint32_t i = 0; i < numThreads; i++)
        hipStreamCreate(&streams[i]);
		//hipStreamCreateWithFlags( &streams[i], hipStreamNonBlocking	);
}

GPUStream::~GPUStream() {

    for (uint32_t i = 0; i < numThreads; i++)
        hipStreamDestroy(streams[i]);

    delete[] streams;
}
